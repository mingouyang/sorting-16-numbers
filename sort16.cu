/* The code is described in this paper: Ouyang M, Sorting sixteen
 * numbers.  Proceedings of IEEE High Performance Extreme Computing
 * Conference (HPEC), 2015, 1-6.
 *
 * Copyright (c) 2015 Ming Ouyang
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use, copy,
 * modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT.  IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define NoOP      0xFFFFFFFFu
#define TwoTo16   0x00010000u
#define BlockSize 256

#define min(x,y) ((x) < (y) ? (x) : (y))
#define max(x,y) ((x) > (y) ? (x) : (y))

int deviceNum = 0;
int dataSize = 16 * TwoTo16;
int *data, *sorted, *gpuData;

void initGPU(void) {
  hipSetDevice(deviceNum);
  hipMalloc((void**) &gpuData, sizeof(int) * dataSize);
}

void init(int argc, char *argv[]) {
  unsigned i, j, k, tmp;
  int c;

  while ((c = getopt(argc, argv, "d:")) != -1)
    switch (c) {
    case 'd':
      sscanf(optarg, "%d", &deviceNum);
      break;
    default:
      break;
    }

  data   = (int*) malloc(sizeof(int) * dataSize);
  sorted = (int*) malloc(sizeof(int) * dataSize);

  j = 0;
  for (i = 0; i < TwoTo16; i++) {
    tmp = i;
    for (k = 0; k < 16; k++) {
      data[j++] = tmp & 0x00000001u;
      tmp = tmp >> 1;
    }
  }

  initGPU();
}

void verify(void) {
  unsigned i, j, k, count;

  count = 0;
  for (i = 0; i < TwoTo16; i++) {
    for (j = 0; j < 15; j++) {
      if (sorted[i * 16 + j] > sorted[i * 16 + j + 1]) {
	printf("not sorted %d:", i);

	for (k = 0; k < 16; k++)
	  printf(" %d", sorted[i * 16 + k]);
	printf("\n");

	if (count++ == 10)
	  exit(1);
      }
    }
  }
}

__device__ inline void IntComparator(int &A, int &B) {
  int t;

  if (A > B) {
    t = A;
    A = B;
    B = t;
  }
}

__device__ inline void UnsignedComparator(unsigned &A, unsigned &B) {
  unsigned t;

  if (A > B) {
    t = A;
    A = B;
    B = t;
  }
}

//Nvidia's implementation of Batcher's sorting network
__global__ void nvidiaBatcher(int *X) {
  __shared__ int sX[BlockSize * 2];
  unsigned pos, size, stride, offset;
  unsigned base = blockIdx.x * BlockSize * 2 + threadIdx.x;

  sX[threadIdx.x] = X[base];
  sX[threadIdx.x + BlockSize] = X[base + BlockSize];
  __syncthreads();

#pragma unroll
  for (size = 2; size <= 16; size <<= 1) {
    stride = size >> 1;
    offset = threadIdx.x & (stride - 1);
    pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
    IntComparator(sX[pos], sX[pos + stride]);
    stride >>= 1;
    for (; stride > 0; stride >>= 1) {
      pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      if (offset >= stride) //divergent computation
	IntComparator(sX[pos - stride], sX[pos]);
    }
  }
  __syncthreads();

  X[base] = sX[threadIdx.x];
  X[base + BlockSize] = sX[threadIdx.x + BlockSize];
}

//divergent computation in Nvidia's implementation is removed
__global__ void newBatcher(int *X) {
  __shared__ int sX[BlockSize * 2];
  unsigned pos, size, stride, offset;
  unsigned base = blockIdx.x * BlockSize * 2 + threadIdx.x;

  sX[threadIdx.x] = X[base];
  sX[threadIdx.x + BlockSize] = X[base + BlockSize];
  __syncthreads();

#pragma unroll
  for (size = 2; size <= 16; size <<= 1) {
    stride = size >> 1;
    offset = threadIdx.x & (stride - 1);
    pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
    IntComparator(sX[pos], sX[pos+stride]);
    stride >>= 1;
    for (; stride > 0; stride >>= 1) {
      pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      //non-divergent computation
      IntComparator(sX[pos - stride * (offset >= stride ? 1 : 0)], sX[pos]);
    }
  }
  __syncthreads();

  X[base] = sX[threadIdx.x];
  X[base + BlockSize] = sX[threadIdx.x + BlockSize];
}

//Van Voorhis's optimal sorting network for 16 numbers
__global__ void VanVoorhis(int *X) {
  __shared__ int sX[BlockSize * 2];
  unsigned wire1, wire2, our16tuple;
  unsigned base = blockIdx.x * BlockSize * 2 + threadIdx.x;

  sX[threadIdx.x] = X[base];
  sX[threadIdx.x + BlockSize] = X[base + BlockSize];
  our16tuple = (threadIdx.x >> 3) << 4;
  __syncthreads();

  //step I
  wire1 = ((threadIdx.x & 7) << 1) + our16tuple;
  wire2 = wire1 + 1;
  IntComparator(sX[wire1], sX[wire2]);

  //step II
  wire1 = (((threadIdx.x & 6) << 1) | (threadIdx.x & 1)) + our16tuple;
  wire2 = wire1 + 2;
  IntComparator(sX[wire1], sX[wire2]);

  //step III
  wire1 = (((threadIdx.x & 4) << 1) | (threadIdx.x & 3)) + our16tuple;
  wire2 = wire1 + 4;
  IntComparator(sX[wire1], sX[wire2]);

  //step IV
  wire1 = (threadIdx.x & 7) + our16tuple;
  wire2 = wire1 + 8;
  IntComparator(sX[wire1], sX[wire2]);

  //step V
  wire1 = threadIdx.x & 7;
  wire1 = (wire1 == 2) ? 13 : wire1;
  wire2 = ((wire1 << 1) & 10) | ((wire1 >> 1) & 5);
  wire2 = (wire1 == wire2) ? (wire2 ^ 15) : wire2;
  wire1 += our16tuple;
  wire2 += our16tuple;
  IntComparator(sX[wire1], sX[wire2]);

  //step VI
  wire1 = threadIdx.x & 7;
  wire2 = ((wire1 << 1) - (wire1 & 1)) << (!(wire1 >> 2));
  wire2 = (wire1 == 0) ? 15 : wire2;
  wire2 = (wire1 == 1) ? 4 : wire2;
  if (wire1 == 4){
    wire1 = wire1 ^ 15;
    wire2 = wire1 + 3;
  }
  wire1 += our16tuple;
  wire2 += our16tuple;
  IntComparator(sX[wire1], sX[wire2]);

  //step VII
  wire1 = (((threadIdx.x & 6) << 1) | (threadIdx.x & 1) | 2) + our16tuple;
  wire2 = (((((threadIdx.x & 6) << 1) | (threadIdx.x & 1) | 2) + 2) & 15)
    + our16tuple;
  UnsignedComparator(wire1, wire2);
  IntComparator(sX[wire1], sX[wire2]);

  //step VIII
  wire1 = ((threadIdx.x & 7) << 1) + 1 + our16tuple;
  wire2 = ((((threadIdx.x & 7) << 1) + 4) & 15) + our16tuple;
  UnsignedComparator(wire1, wire2);
  IntComparator(sX[wire1], sX[wire2]);

  //step IX
  wire1 = ((threadIdx.x & 7) << 1) + 1 + our16tuple;
  wire2 = ((((threadIdx.x & 7) << 1) + 2) & 15) + our16tuple;
  UnsignedComparator(wire1, wire2);
  IntComparator(sX[wire1], sX[wire2]);

  __syncthreads();
  X[base] = sX[threadIdx.x];
  X[base + BlockSize] = sX[threadIdx.x + BlockSize];
}

int main(int argc, char *argv[]) {
  hipEvent_t start;
  hipEvent_t stop;
  float msec;

  init(argc, argv);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(gpuData, data, sizeof(int) * dataSize, hipMemcpyHostToDevice);
  hipEventRecord(start, NULL);
  nvidiaBatcher <<<dataSize / (BlockSize * 2), BlockSize>>> (gpuData);
  hipDeviceSynchronize();
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msec, start, stop);
  printf("Nvd Batcher, blocksize %d, device %d, %.3f ms\n",
	 BlockSize, deviceNum, msec);
  hipMemcpy(sorted, gpuData, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
  verify();

  hipMemcpy(gpuData, data, sizeof(int) * dataSize, hipMemcpyHostToDevice);
  hipEventRecord(start, NULL);
  VanVoorhis <<<dataSize / (BlockSize * 2), BlockSize>>> (gpuData);
  hipDeviceSynchronize();
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msec, start, stop);
  printf("Van Voorhis, blocksize %d, device %d, %.3f ms\n",
	 BlockSize, deviceNum, msec);
  hipMemcpy(sorted, gpuData, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
  verify();

  hipMemcpy(gpuData, data, sizeof(int) * dataSize, hipMemcpyHostToDevice);
  hipEventRecord(start, NULL);
  newBatcher <<<dataSize / (BlockSize * 2), BlockSize>>> (gpuData);
  hipDeviceSynchronize();
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msec, start, stop);
  printf("new Batcher, blocksize %d, device %d, %.3f ms\n",
	 BlockSize, deviceNum, msec);
  hipMemcpy(sorted, gpuData, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
  verify();

  return 0;
}
